/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   printf("Hello world from thread %d, Name: Abner Garcia, Carnet: 21285\n", idx);

   if (idx == 1023) {
    printf("Thread 1023, Name: Abner Garcia, Carnet: 21285\n");
   }
}

int main()
{
  // Configurar 2 bloques de 1024 hilos cada uno
  hello<<<2, 1024>>>();
  hipDeviceSynchronize(); 
  return 0;
}