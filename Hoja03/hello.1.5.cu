/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello.cu -o hello -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello()
{
   int idx = blockIdx.x * blockDim.x + threadIdx.x;
   printf("Hello world from thread %d, Name: Abner Garcia, Carnet: 21285\n", idx);

   if (idx == 2047) {
    printf("Thread 2047, Name: Abner Garcia, Carnet: 21285\n");
   }
}

int main()
{
  // Configurar 1 bloque de 2048 hilos
  hello<<<1, 2048>>>();
  hipDeviceSynchronize(); 
  return 0;
}