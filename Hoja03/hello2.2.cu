#include "hip/hip_runtime.h"
/*
 ============================================================================
 Author        : G. Barlas
 Version       : 1.0
 Last modified : December 2014
 License       : Released under the GNU GPL 3.0
 Description   :
 To build use  : nvcc hello2.cu -o hello2 -arch=sm_20
 ============================================================================
 */
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello ()
{
  int myID = (blockIdx.z * gridDim.x * gridDim.y  +
              blockIdx.y * gridDim.x +
              blockIdx.x ) * blockDim.x * blockDim.y * blockDim.z +
              threadIdx.z *  blockDim.x * blockDim.y +
              threadIdx.y * blockDim.x +
              threadIdx.x;

//  Simplification of above
  //grid: 3D --- z,y,x: all dims and blockids
  //block: 1D -- x
  //int myID = ( blockIdx.z * gridDim.x * gridDim.y  +
  //             blockIdx.y * gridDim.x +
  //             blockIdx.x ) * blockDim.x +
  //             threadIdx.x;

  // Obtén el número total de hilos en la grilla
  int totalThreads = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;

  // Comprueba si este hilo tiene el ID global máximo
  if (myID == totalThreads - 1)
  {
    printf ("Hello world from %i by Abner Garcia - 21285\n", myID);
  }
}

int main ()
{
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);  // Obtener las propiedades de la GPU 0

  printf("Max Threads Per Block (x): %d\n", prop.maxThreadsDim[0]); //1024
  printf("Max Threads Per Block (y): %d\n", prop.maxThreadsDim[1]); //1024
  printf("Max Threads Per Block (z): %d\n", prop.maxThreadsDim[2]); //64
  printf("Max Blocks Per Grid (x): %d\n", prop.maxGridSize[0]); // 2147483647
  printf("Max Blocks Per Grid (y): %d\n", prop.maxGridSize[1]); // 65535
  printf("Max Blocks Per Grid (z): %d\n", prop.maxGridSize[2]); // 65535
  // Definir el número total de datos
  int maxThreadsX = 1024;
  int maxThreadsY = 1024;
  int maxThreadsZ = 64;
  int maxBlocksX = 2147483647;
  int maxBlocksY = 65535;
  int maxBlocksZ = 65535;
  int totalData = 100000;
  // obtener el número total de hilos en la grilla y el número de hilos por bloque
  dim3 threadsPerBlock(
    min(totalData, maxThreadsX),
    min(1, maxThreadsY),          
    min(1, maxThreadsZ)
  );
  // obtener el número de bloques por grilla y el número de bloques por eje
  dim3 blocksPerGrid(
    min((totalData + threadsPerBlock.x - 1) / threadsPerBlock.x, maxBlocksX),
    min(1, maxBlocksY),
    min(1, maxBlocksZ)
  );

  hello <<< blocksPerGrid, threadsPerBlock >>> ();
  hipDeviceReset();

  return 0;
}